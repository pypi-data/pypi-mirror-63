#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "dispatch.h"
#include "elementwise.cuh"
#include "saturate_cast.cuh"


namespace augpy {


template<typename scalar_t>
__device__ __forceinline__ void copy_function(
        array<tensor_param, 2> tensors,
        unsigned char nothing
){
    *reinterpret_cast<scalar_t*>(tensors[0].ptr) = *reinterpret_cast<scalar_t*>(tensors[1].ptr);
}


CudaTensor* copy(
        CudaTensor* src,
        CudaTensor* dst,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(dst, src);
    DISPATCH(src->dl_tensor.dtype, "copy", ([&] {
        retval = elementwise_function<2, unsigned char, copy_function<scalar_t>>(
            tensors, 0, blocks_per_sm, num_threads
        );
    }));
    return retval;
}


template<typename scalar_t>
__device__ __forceinline__ void fill_function(
        array<tensor_param, 1> tensors,
        scalar_t fill_value
){
    *reinterpret_cast<scalar_t*>(tensors[0].ptr) = fill_value;
}


void fill(
        double scalar,
        CudaTensor* dst,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    auto tensors = make_array(dst);
    DISPATCH(dst->dl_tensor.dtype, "fill", ([&] {
        scalar_t casted_scalar;
        saturate_cast<double, scalar_t>(scalar, &casted_scalar);
        elementwise_function<1, scalar_t, fill_function<scalar_t>>(
            tensors, casted_scalar, blocks_per_sm, num_threads
        );
    }));
}


// namespace augpy
}
