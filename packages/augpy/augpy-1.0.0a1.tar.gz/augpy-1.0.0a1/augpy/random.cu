#include "hip/hip_runtime.h"
#include <limits>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "core.h"
#include "tensor.h"
#include "saturate_cast.cuh"
#include "exception.h"
#include "dispatch.h"
#include "random.h"
#include <random>

#define EPSILON_FLOAT 1e-7
#define EPSILON_DOUBLE 1e-15
namespace augpy {


RandomNumberGenerator::RandomNumberGenerator(py::object* device_id_, py::object* seed_) {
    int new_device_id;
    int* device_id_p = NULL;
    unsigned long long new_seed;
    unsigned long long* seed_p = NULL;
    if (device_id_ && !device_id_->is_none()) {
        new_device_id = device_id_->cast<int>();
        device_id_p = &new_device_id;
    }
    if (seed_ && !seed_->is_none()) {
        new_seed = seed_->cast<unsigned long long>();
        seed_p = &new_seed;
    }
    init_device_state(device_id_p, seed_p);
}


RandomNumberGenerator::RandomNumberGenerator(int* device_id, unsigned long long* seed) {
    init_device_state(device_id, seed);
}


RandomNumberGenerator::~RandomNumberGenerator() noexcept(false) {
    if (device_states != NULL) {
        try {
            managed_cudafree(device_states);
        }
        // exceptions in destructor are not forwarded to Python,
        // so we cannot throw here :(
        catch (cnmem_error& e) {}
    }
}


template <typename rng_t>
__global__ void initialize_states_kernel(
    rng_t *state,
    size_t num_states,
    unsigned long long seed
) {
    unsigned long long sequence = blockDim.x * blockIdx.x + threadIdx.x;
    if (sequence < num_states) {
        hiprand_init(seed, sequence, 0, &state[sequence]);
    }
}


void RandomNumberGenerator::init_device_state(int* device_id_, unsigned long long* seed_) {
    // get default values
    if (!device_id_) {
        device_id_ = &current_device;
    }
    this->device_id = *device_id_;
    this->seed = 0;
    if (seed_) {
        this->seed = *seed_;
    }
    else {
        std::random_device initial;
        int steps = sizeof(unsigned long long) / sizeof(unsigned int);
        for (int i=0; i<steps; ++i) {
            seed <<= sizeof(unsigned int);
            seed += initial();
        }
    }

    // initialize state
    CUDA(hipSetDevice(device_id));
    hipDeviceProp_t props = get_device_properties(device_id);
    num_states = get_num_cuda_cores(device_id);
    size_t size = num_states * sizeof(rng_t);
    managed_cudamalloc((void **)&device_states, size, device_id);
    size_t num_blocks = ceil_div(num_states, 128);

    // retrieve the current stack limit, i.e.,
    // the number of bytes the stack of each thread can hold
    size_t stacklimit;
    CUDA(hipDeviceGetLimit(&stacklimit, hipLimitStackSize));

    // calls to hiprand_init may require a very large stack size,
    // thus increasing the reserved space in device memory
    initialize_states_kernel<rng_t><<<num_blocks, 128, 0, current_stream>>>(
        device_states, num_states, seed
    );

    // wait for random state initialization to finish
    // and reset stack limit back to previous value
    CUDA(hipStreamSynchronize(current_stream));
    CUDA(hipDeviceSetLimit(hipLimitStackSize, stacklimit));
}


template<typename rng_t, typename scalar_t>
__device__ __forceinline__ scalar_t __curand_uniform(rng_t* state);

template<>
__device__ __forceinline__ float __curand_uniform(rng_t* state){
    return hiprand_uniform(state);
}

template<>
__device__ __forceinline__ double __curand_uniform(rng_t* state){
    return hiprand_uniform_double(state);
}


template<typename rng_t, typename scalar_t, typename temp_t>
__global__ void generate_uniform_kernel(
    scalar_t* result,
    const size_t n,
    const unsigned int m,
    const temp_t offset,
    const temp_t scale,
    rng_t* states,
    const int num_states
) {
    unsigned int i = (blockDim.x * blockIdx.x * m + threadIdx.x);
    unsigned int state_i = (blockDim.x * blockIdx.x + threadIdx.x) % num_states;
    size_t max_i = min(((size_t)i)+blockDim.x*m, n);
    rng_t state = states[state_i];
    for (unsigned int j=i; j<max_i; j+=blockDim.x) {
        result[j] = (scalar_t) (
            // hiprand_uniform produces floats x \in (0, 1]
            // use 1-x through offset and scale:
            // scale = vmax - vmin
            // offset = vmin + scale
            offset - scale * __curand_uniform<rng_t, temp_t>(&state)
        );
    }
    states[state_i] = state;
}


void RandomNumberGenerator::uniform(
        CudaTensor* target,
        double vmin,
        double vmax,
        unsigned int blocks_per_sm,
        unsigned int threads
) {
    //init_device_state();

    py::gil_scoped_release release;

    assert_contiguous(target);

    DLTensor &t = target->dl_tensor;

    if (t.ctx.device_id != device_id) {
        throw std::invalid_argument("tensor exceeds max size");
    }

    calc_threads(threads, device_id);
    size_t num;
    unsigned int num_blocks;
    unsigned int values_per_thread = 0;
    calc_blocks_values_1d(t, num_blocks, num, values_per_thread, threads, blocks_per_sm);

    if (num == 0) {
        return;
    }

    if (t.dtype.code == kDLInt || t.dtype.code == kDLUInt  ) {
        vmax += 1;
        if(t.dtype.bits <= 16){
            vmin *= 1+EPSILON_FLOAT;
            vmax *= 1-EPSILON_FLOAT;
        }
        else {
            vmin *= 1+EPSILON_DOUBLE;
            vmax *= 1-EPSILON_DOUBLE;
        }
    }
    double scale = vmax - vmin;
    double offset = vmin + scale;
    DISPATCH(t.dtype, "generate_uniform_kernel", ([&] {
        generate_uniform_kernel<rng_t, scalar_t, temp_t>
        <<<num_blocks, threads, 0, current_stream>>>(
            (scalar_t*) target->ptr(),
            num,
            values_per_thread,
            (temp_t) offset,
            (temp_t) scale,
            device_states,
            num_states
        );
    }));
    CUDA(hipGetLastError());
    // mark tensors as in use
    target->record();
}


template<typename scalar_t>
__device__ __forceinline__ void __generate_normal2(
        scalar_t* result, unsigned int j, dim3 blockDim, rng_t* state,
        float mean, float std
) {
    float2 v = hiprand_normal2(state);
    saturate_cast<float, scalar_t>(std * v.x + mean, &result[j]);
    saturate_cast<float, scalar_t>(std * v.y + mean, &result[j+blockDim.x]);
}

template<typename scalar_t>
__device__ __forceinline__ void __generate_normal2(
        scalar_t* result, unsigned int j, dim3 blockDim, rng_t* state,
        double mean, double std
) {
    double2 v = hiprand_normal2_double(state);
    saturate_cast<double, scalar_t>(std * v.x + mean, &result[j]);
    saturate_cast<double, scalar_t>(std * v.y + mean, &result[j+blockDim.x]);
}

template<typename scalar_t>
__device__ __forceinline__ void __generate_normal(
        scalar_t* result, unsigned int j, rng_t* state,
        float mean, float std
) {
    saturate_cast<float, scalar_t>(std * hiprand_normal(state) + mean, &result[j]);
}

template<typename scalar_t>
__device__ __forceinline__ void __generate_normal(
        scalar_t* result, unsigned int j, rng_t* state,
        double mean, double std
) {
    saturate_cast<double, scalar_t>(std * hiprand_normal(state) + mean, &result[j]);
}


template<typename rng_t, typename scalar_t, typename temp_t>
__global__ void generate_gaussian_kernel(
    scalar_t* result,
    const ssize_t n,
    const unsigned int m,
    const temp_t mean,
    const temp_t std,
    rng_t* states,
    const unsigned int num_states
) {
    unsigned int i = (blockDim.x * blockIdx.x * m + threadIdx.x);
    size_t max_i = min(((size_t)i) + blockDim.x * m, n - blockDim.x);
    unsigned int state_i = (blockDim.x * blockIdx.x + threadIdx.x) % num_states;
    rng_t state = states[state_i];
    unsigned int j;
    for(j=i; j<max_i; j+=2*blockDim.x) {
        __generate_normal2<scalar_t>(result, j, blockDim, &state, mean, std);
    }
    if (j < n) {
        __generate_normal<scalar_t>(result, j, &state, mean, std);
    }
    states[state_i] = state;
}


void RandomNumberGenerator::gaussian(
        CudaTensor* target,
        double mean,
        double std,
        unsigned int blocks_per_sm,
        unsigned int threads
) {
    //init_device_state();

    py::gil_scoped_release release;

    assert_contiguous(target);

    DLTensor &t = target->dl_tensor;

    if (t.ctx.device_id != device_id) {
        throw std::invalid_argument("tensor must be on same device as generator");
    }

    calc_threads(threads, device_id);
    size_t num;
    unsigned int num_blocks;
    unsigned int values_per_thread = 0;
    calc_blocks_values_1d(t, num_blocks, num, values_per_thread, threads, blocks_per_sm);

    if (num == 0) {
        return;
    }

    DLDataType dtype = target->dl_tensor.dtype;
    DISPATCH(dtype, "generate_gaussian_kernel", ([&] {
        generate_gaussian_kernel<rng_t, scalar_t, temp_t>
        <<<num_blocks, threads, 0, current_stream>>>(
            (scalar_t*) target->ptr(),
            num,
            values_per_thread,
            (temp_t) mean,
            (temp_t) std,
            device_states,
            num_states
        ); })
    );
    CUDA(hipGetLastError());
    // mark tensors as in use
    target->record();
}


// namespace augpy
}
