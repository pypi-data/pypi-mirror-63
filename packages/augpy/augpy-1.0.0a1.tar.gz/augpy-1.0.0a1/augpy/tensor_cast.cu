#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "dispatch.h"
#include "saturate_cast.cuh"
#include "elementwise.cuh"
#include <tuple>


namespace augpy {


__device__ __forceinline__ void cast_function(
        array<tensor_param, 2> tensors,
        std::tuple<DLDataType, DLDataType> dtypes
){
    __DISPATCH_NOEXC(std::get<0>(dtypes), dst_t, "cast", ([&] {
        __DISPATCH_NOEXC(std::get<1>(dtypes), src_t, "cast", ([&] {
            saturate_cast<src_t, dst_t>(
                *reinterpret_cast<src_t*>(tensors[1].ptr),
                reinterpret_cast<dst_t*>(tensors[0].ptr)
            );
        }));
    }));
}


void cast_tensor(
        CudaTensor* src,
        CudaTensor* dst,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    if (!src || !dst) {
        throw std::invalid_argument("source and result tensor need to be a valid tensors");
    }
    DLTensor &t_src = src->dl_tensor;
    DLTensor &t_dst = dst->dl_tensor;
    if (t_src.ndim != t_dst.ndim || !array_equals(0, t_src.ndim, t_src.shape, t_dst.shape)) {
        throw std::invalid_argument("source and result tensor must have same shape");
    }
    auto tensors = make_array(dst, src);
    auto dtypes = std::make_tuple(t_dst.dtype, t_src.dtype);
    elementwise_function<2, std::tuple<DLDataType, DLDataType>, cast_function>(
        tensors, dtypes, blocks_per_sm, num_threads, false
    );
}


CudaTensor* cast_type(
        CudaTensor* tensor,
        DLDataType dtype,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    DLTensor &t = tensor->dl_tensor;
    CudaTensor* out = new CudaTensor(&t.shape[0], t.ndim, dtype, t.ctx.device_id);
    cast_tensor(tensor, out, blocks_per_sm, num_threads);
    return out;
}


// namespace augpy
}
