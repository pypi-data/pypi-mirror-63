#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "dispatch.h"
#include "saturate_cast.cuh"
#include "elementwise.cuh"
#include "cub/device/device_reduce.cuh"
#include "cub/iterator/transform_input_iterator.cuh"


namespace augpy {


// logical AND operator
template<typename temp_t>
struct SaturatedSum
{
    template <typename scalar_t>
    __device__ __forceinline__
    scalar_t operator()(const scalar_t &a, const scalar_t &b) const {
        scalar_t result;
        saturate_cast<temp_t, scalar_t>((temp_t)a + (temp_t)b, &result);
        return result;
    }
};


CudaTensor* sum(
        CudaTensor* tensor,
        bool upcast
){
    if (!tensor->is_contiguous()) {
        throw std::invalid_argument("input must be contiguous");
    }
    DLTensor& t = tensor->dl_tensor;
    size_t num_items = numel(t);
    size_t temp_storage_bytes;
    CudaTensor* temp=nullptr;
    CudaTensor* out=nullptr;
    hipError_t e;
    DISPATCH(t.dtype, "sum_kernel", ([&] {
        // interator that casts to temp_t
        hipcub::CastOp<temp_t> op_cast;
        hipcub::TransformInputIterator<double, hipcub::CastOp<temp_t>, scalar_t*> it_cast(
            (scalar_t*)tensor->ptr(), op_cast
        );
        // get required temp storage size
        CUDA(hipcub::DeviceReduce::Sum(
            nullptr, temp_storage_bytes,
            (scalar_t*) nullptr, (temp_t*) nullptr,
            num_items
        ));
        int64_t tsb = (int64_t) temp_storage_bytes;
        temp = new CudaTensor(&tsb, 1, dldtype_uint8, current_device);
        // sum with upcast to temp_t
        out = new CudaTensor(nullptr, 0, get_dldatatype<temp_t>(), current_device);
        e = hipcub::DeviceReduce::Sum(
            temp->ptr(),
            temp_storage_bytes,
            it_cast,
            (temp_t*) out->ptr(),
            num_items,
            current_stream
        );
    }));
    if (e==hipSuccess) {
        temp->record();
    }
    if (temp) {
        delete temp;
    }
    if (e!=hipSuccess) {
        if (out) {
            delete out;
        }
        CUDA(e);
    }
    if (!upcast) {
        temp = cast_type(out, t.dtype);
        delete out;
        out = temp;
    }
    return out;
}


template <typename scalar_t, typename temp_t>
__device__ __forceinline__ void __sum_axis_sat(
        array<tensor_param, 2> tensors,
        std::tuple<int64_t, int64_t> params
){
    temp_t v = 0;
    for(int64_t count=0; count<std::get<0>(params); ++count) {
        v += (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr);
        tensors[1].ptr += std::get<1>(params);
    }
    saturate_cast<temp_t, scalar_t>(v, reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


template <typename scalar_t, typename temp_t>
__device__ __forceinline__ void __sum_axis_cast(
        array<tensor_param, 2> tensors,
        std::tuple<int64_t, int64_t> params
){
    temp_t v = 0;
    for(int64_t count=0; count<std::get<0>(params); ++count) {
        v += (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr);
        tensors[1].ptr += std::get<1>(params);
    }
    *reinterpret_cast<temp_t*>(tensors[0].ptr) = v;
}


CudaTensor* sum_axis(
        CudaTensor* tensor,
        int axis,
        bool keepdim,
        bool upcast,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    DLTensor& t = tensor->dl_tensor;
    if (axis < 0 || axis >= t.ndim) {
        throw std::invalid_argument("axis out of range");
    }
    int64_t new_shape[DLTENSOR_MAX_NDIM];
    int64_t new_strides[DLTENSOR_MAX_NDIM];
    int new_ndim = 0;
    for (int dim=0; dim<t.ndim; ++dim) {
        if (dim == axis) {
            if (keepdim) {
                new_shape[new_ndim] = 1;
                new_strides[new_ndim] = 1;
                ++new_ndim;
            }
        }
        else {
            new_shape[new_ndim] = t.shape[dim];
            new_strides[new_ndim] = t.strides[dim];
            ++new_ndim;
        }
    }
    CudaTensor tensor_noaxis(tensor, new_ndim, new_shape, new_strides, 0);
    CudaTensor* retval = nullptr;
    DISPATCH(tensor->dl_tensor.dtype, "sum_axis", ([&] {
        // sum with upcast to temp_t
        if (upcast) {
            if (!result) {
                retval = result = new CudaTensor(
                    tensor_noaxis.dl_tensor.shape,
                    tensor_noaxis.dl_tensor.ndim,
                    get_dldatatype<temp_t>(),
                    tensor_noaxis.dl_tensor.ctx.device_id
                );
            }
            elementwise_function<2, std::tuple<int64_t, int64_t>, __sum_axis_cast<scalar_t, temp_t>>(
                make_array(result, &tensor_noaxis),
                std::make_tuple(t.shape[axis], t.strides[axis] * itemsize(t.dtype)),
                blocks_per_sm, num_threads, false
            );
        }
        // saturating sum
        else {
            if (!result) {
                retval = result = new CudaTensor(
                    tensor_noaxis.dl_tensor.shape,
                    tensor_noaxis.dl_tensor.ndim,
                    get_dldatatype<scalar_t>(),
                    tensor_noaxis.dl_tensor.ctx.device_id
                );
            }
            elementwise_function<2, std::tuple<int64_t, int64_t>, __sum_axis_sat<scalar_t, temp_t>>(
                make_array(result, &tensor_noaxis),
                std::make_tuple(t.shape[axis], t.strides[axis] * itemsize(t.dtype)),
                blocks_per_sm, num_threads, false
            );
        }

    }));
    return retval;
}


// logical AND operator
struct AndOp
{
    template <typename T>
    __device__ __forceinline__
    uint8_t operator()(const T &a, const T &b) const {
        return a && b;
    }
};


CudaTensor* all(CudaTensor* tensor) {
    DLTensor& t = tensor->dl_tensor;
    size_t num_items = numel(t);
    size_t temp_storage_bytes;
    CudaTensor* temp=nullptr;
    CudaTensor* out=nullptr;
    // call CUB
    hipError_t e;
    AndOp op;
    DISPATCH(t.dtype, "all_kernel", ([&] {
        // get required temp storage size
        CUDA(hipcub::DeviceReduce::Reduce(
            nullptr, temp_storage_bytes,
            (scalar_t*) nullptr, (scalar_t*) nullptr,
            num_items, op, (uint8_t) 1
        ));
        // create temp storage and output
        int64_t tsb = (int64_t) temp_storage_bytes;
        temp = new CudaTensor(&tsb, 1, dldtype_uint8, current_device);
        out = new CudaTensor(nullptr, 0, dldtype_uint8, current_device);
        e = hipcub::DeviceReduce::Reduce(
            temp->ptr(),
            temp_storage_bytes,
            (scalar_t*) tensor->ptr(),
            (scalar_t*) out->ptr(),
            num_items,
            op,
            (uint8_t) 1,
            current_stream
        );
    }));
    if (e==hipSuccess) {
        temp->record();
    }
    if (temp) {
        delete temp;
    }
    if (e!=hipSuccess) {
        if (out) {
            delete out;
        }
        CUDA(e);
    }
    return out;
}


// namespace augpy
}
