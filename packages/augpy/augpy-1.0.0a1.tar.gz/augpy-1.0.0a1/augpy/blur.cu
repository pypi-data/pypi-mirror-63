#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "core.h"
#include "tensor.h"
#include "saturate_cast.cuh"
#include "blur.h"
#include "dispatch.h"
#include "exception.h"


namespace augpy {


#define clamp(x,a,b) min(max(x,a),b)
#define BLOCKSIZE 16
#define MAX_SMEM_PER_BLOCK 48*1024
#define SQRT_TWO_PI 2.50662827463
#define SQRT_TWO 1.41421356237


template <typename scalar_t, typename temp_t>
__global__ void box_blur_single_kernel(
        scalar_t* image,
        scalar_t* target,
        const int ksize,
        const int offset,
        const temp_t norm,
        const int C,
        const int H,
        const int W,
        const ssize_t image_channel_stride,
        const ssize_t target_channel_stride
) {
    extern __shared__ char smem[];

    const int source_x = blockIdx.y * blockDim.x + threadIdx.x;
    const int source_y = blockIdx.z * blockDim.y + threadIdx.y;

    target += source_y*W + source_x;

    const int shared_w = blockDim.x + ksize - 1;
    const int shared_h = blockDim.y + ksize - 1;
    temp_t* horizontal_data = reinterpret_cast<temp_t*>(smem);
    temp_t* vertical_data = &horizontal_data[shared_h * shared_w];
    const int first_i = threadIdx.y * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * blockDim.y;

    const int maxx = W-1;
    const int maxy = H-1;
    const int offset_x = (int)(blockIdx.y * blockDim.x) - offset;
    const int offset_y = (int)(blockIdx.z * blockDim.y) - offset;
    for(int c=0; c<C; c++){
        for(int i=first_i; i<shared_h*shared_w ; i+=num_threads){
            int x = clamp(i % shared_w + offset_x, 0, maxx);
            int y = clamp(i / shared_w + offset_y, 0, maxy);
            horizontal_data[i] = image[y*W + x];
        }

        __syncthreads();

        for(int i=first_i; i<shared_h*blockDim.x; i+=num_threads) {
            int x = i % blockDim.x;
            int y = i / blockDim.x;
            temp_t sum = 0;
            for(int k=0; k<ksize; k++) {
                sum += horizontal_data[y*shared_w + x + k];
            }
            vertical_data[i] = sum * norm;
        }

        __syncthreads();

        if(source_x >= W || source_y >= H){
            image += image_channel_stride;
            target += target_channel_stride;
            continue;
        }

        temp_t sum = 0;
        for(int k=0; k<ksize; k++) {
            sum += vertical_data[(threadIdx.y + k) * blockDim.x + threadIdx.x];
        }
        saturate_cast<temp_t, scalar_t>(sum * norm, target);
        image += image_channel_stride;
        target += target_channel_stride;
    }
}


CudaTensor* box_blur_single(
        CudaTensor* input,
        int ksize,
        CudaTensor* out
) {
    CudaTensor* retval = NULL;
    if (!out) {
        out = retval = empty_like(input);
    }

    assert_contiguous(input);
    assert_contiguous(out);

    DLTensor &input_tensor = input->dl_tensor;
    DLTensor &out_tensor = out->dl_tensor;

    if(input_tensor.ndim != 3 || out_tensor.ndim != 3) {
        throw std::invalid_argument("need 3D input and output tensors");
    }

    int C = input_tensor.shape[0];
    int H = input_tensor.shape[1];
    int W = input_tensor.shape[2];

    if(!array_equals(0, 3, input_tensor.shape, out_tensor.shape)) {
        throw std::invalid_argument("input and output shapes need to be equal");
    }

    const int shared_size = BLOCKSIZE + ksize - 1;
    int shared_memory;
    dim3 grid_dim(1, (W+BLOCKSIZE-1)/BLOCKSIZE, (H+BLOCKSIZE-1)/BLOCKSIZE);
    dim3 block_dim(BLOCKSIZE, BLOCKSIZE, 1);

    DISPATCH(input_tensor.dtype, "box_blur_single_kernel", ([&] {
        shared_memory = ksize * sizeof(temp_t)
                      + shared_size * shared_size * sizeof(temp_t)
                      + BLOCKSIZE * shared_size * sizeof(temp_t);

        if(shared_memory > MAX_SMEM_PER_BLOCK) {
            throw std::invalid_argument("kernel size too large.");
        }
        box_blur_single_kernel<scalar_t, temp_t>
        <<<grid_dim, block_dim, shared_memory, current_stream>>>(
            (scalar_t*)input->ptr(),
            (scalar_t*)out->ptr(),
            ksize,
            ksize / 2,
            1.F / (float) ksize,
            C, H, W, H*W, H*W
        );
    }));
    CUDA(hipGetLastError());
    input->record();
    out->record();

    return retval;
}


template <typename temp_t>
__device__ __forceinline__ void create_gaussian_kernel(
        temp_t* kernel,
        float sigma,
        int ksize,
        int range
) {
    if(threadIdx.y * blockDim.x + threadIdx.x < range+1){
        int k = threadIdx.y * blockDim.x + threadIdx.x;
        kernel[k] = exp(-(temp_t)(k * k) / (2 * sigma * sigma));
        temp_t scale = kernel[0];
        for(int s=1; s<range+1; s++) {
            scale += 2 * kernel[s];
        }
        kernel[k] /= scale;
    }
}


template <typename scalar_t, typename temp_t>
__device__ __forceinline__ void gaussian_blur_separable(
        scalar_t* image,
        scalar_t* target,
        temp_t* kernel,
        const int ksize,
        const int range,
        const int source_x,
        const int source_y,
        const int C,
        const int H,
        const int W,
        const ssize_t image_channel_stride,
        const ssize_t target_channel_stride
) {
    const int shared_w = blockDim.x + ksize - 1;
    const int shared_h = blockDim.y + ksize - 1;
    temp_t* horizontal_data = &kernel[range+1];
    temp_t* vertical_data = &horizontal_data[shared_h * shared_w];
    const int first_i = threadIdx.y * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * blockDim.y;
    const int maxx = W-1;
    const int maxy = H-1;
    const int offset_x = (int)(blockIdx.y * blockDim.x) - range;
    const int offset_y = (int)(blockIdx.z * blockDim.y) - range;

    for(int c=0; c<C; c++){
        for(int i=first_i; i<shared_h*shared_w ; i+=num_threads){
            int x = clamp(i % shared_w + offset_x, 0, maxx);
            int y = clamp(i / shared_w + offset_y, 0, maxy);
            horizontal_data[i] = image[y*W + x];
        }

        __syncthreads();

        for(int i=first_i; i<shared_h*blockDim.x; i+=num_threads) {
            int x = i % blockDim.x;
            int y = i / blockDim.x;
            temp_t sum = 0;
            for(int k=0; k<ksize; k++) {
                sum += kernel[abs(k-range)] * horizontal_data[y*shared_w + x + k];
            }
            vertical_data[i] = sum;
        }

        __syncthreads();

        if(source_x >= W || source_y >= H){
            image += image_channel_stride;
            target += target_channel_stride;
            continue;
        }

        temp_t sum = 0;
        for(int k=0; k<ksize; k++) {
            sum += kernel[abs(k-range)] * vertical_data[(threadIdx.y + k) * blockDim.x + threadIdx.x];
        }
        
        saturate_cast<temp_t, scalar_t>(sum, target);

        image += image_channel_stride;
        target += target_channel_stride;
    }
}


template <typename scalar_t, typename temp_t>
__global__ void gaussian_blur_kernel(
        scalar_t* image,
        scalar_t* target,
        const float* sigmas,
        const int max_ksize,
        const int C,
        const int H,
        const int W,
        const ssize_t image_image_stride,
        const ssize_t image_channel_stride,
        const ssize_t target_image_stride,
        const ssize_t target_channel_stride
) {
    extern __shared__ char smem[];
    temp_t* kernel = reinterpret_cast<temp_t*>(smem);
    const int source_x = blockIdx.y * blockDim.x + threadIdx.x;
    const int source_y = blockIdx.z * blockDim.y + threadIdx.y;

    image += blockIdx.x * image_image_stride;
    target += blockIdx.x * target_image_stride + source_y*W + source_x;

    const float sigma = sigmas[blockIdx.x];
    int ksize = (int)(sigma * 6.6f - 2.3f);
    // required kernel size is <= 1, so blur would have no effect
    if(ksize <= 1) {
        if(source_x >= W || source_y >= H) {
            return;
        }
        image += source_y*W + source_x;
        for (int c=0; c<C; c++) {
            *target = *image;
            image += image_channel_stride;
            target += target_channel_stride;
        } 
        return;
    }
    ksize = min(max_ksize, max(3, ksize | 1));
    const int range = ksize / 2;

    create_gaussian_kernel(kernel, sigma, ksize, range);
    gaussian_blur_separable(
        image, target, kernel, ksize, range, source_x, source_y, C, H, W,
        image_channel_stride, target_channel_stride
    );
}


CudaTensor* gaussian_blur(
        CudaTensor* input,
        CudaTensor* sigmas,
        int max_ksize,
        CudaTensor* out
) {
    CudaTensor* retval = NULL;
    if (!out) {
        out = retval = empty_like(input);
    }

    assert_contiguous(input);
    assert_contiguous(sigmas);
    assert_contiguous(out);

    DLTensor &input_tensor = input->dl_tensor;   
    DLTensor &out_tensor = out->dl_tensor;
    DLTensor &sigmas_tensor = sigmas->dl_tensor;

    if(sigmas_tensor.dtype.code != kDLFloat || sigmas_tensor.dtype.bits != 32) {
        throw std::invalid_argument("sigmas must be float32");
    }

    if(input_tensor.ndim != 4 || out_tensor.ndim != 4) {
        throw std::invalid_argument("need 4D input and output tensors");
    }

    int N = input_tensor.shape[0];
    int C = input_tensor.shape[1];
    int H = input_tensor.shape[2];
    int W = input_tensor.shape[3];

    if(!array_equals(0, 4, input_tensor.shape, out_tensor.shape)) {
        throw std::invalid_argument("input and output shapes need to be equal");
    }

    max_ksize = max(3, max_ksize | 1);
    const int shared_size = BLOCKSIZE + max_ksize - 1;
    int shared_memory;
    dim3 grid_dim(N, (W+BLOCKSIZE-1)/BLOCKSIZE, (H+BLOCKSIZE-1)/BLOCKSIZE);
    dim3 block_dim(BLOCKSIZE, BLOCKSIZE, 1);

    DISPATCH(input_tensor.dtype, "gaussian_blur_kernel", ([&] {
        shared_memory = (max_ksize + shared_size * shared_size
                         + BLOCKSIZE * shared_size) * sizeof(temp_t);
        if(shared_memory > MAX_SMEM_PER_BLOCK) {
            throw std::invalid_argument("sigma must be <= 30");
        }
        gaussian_blur_kernel<scalar_t, temp_t>
        <<<grid_dim, block_dim, shared_memory, current_stream>>>(
            (scalar_t*)input->ptr(),
            (scalar_t*)out->ptr(),
            (float*)sigmas->ptr(), max_ksize,
            C, H, W, C*H*W, H*W, C*H*W, H*W
        );
    }));
    CUDA(hipGetLastError());

    input->record();
    sigmas->record();
    out->record();

    return retval;
}


template <typename scalar_t, typename temp_t>
__global__ void gaussian_blur_single_kernel(
        scalar_t* image,
        scalar_t* target,
        const float sigma,
        const int ksize,
        const int range,
        const int C,
        const int H,
        const int W,
        const ssize_t image_channel_stride,
        const ssize_t target_channel_stride
) {
    extern __shared__ char smem[];
    temp_t* kernel = reinterpret_cast<temp_t*>(smem);

    const int source_x = blockIdx.y * blockDim.x + threadIdx.x;
    const int source_y = blockIdx.z * blockDim.y + threadIdx.y;
    target += source_y*W + source_x;

    create_gaussian_kernel(kernel, sigma, ksize, range);
    gaussian_blur_separable(
        image, target, kernel, ksize, range, source_x, source_y, C, H, W,
        image_channel_stride, target_channel_stride
    );
}


CudaTensor* gaussian_blur_single(
        CudaTensor* input,
        float sigma,
        CudaTensor* out
) {
    CudaTensor* retval = NULL;
    if (!out) {
        out = retval = empty_like(input);
    }

    assert_contiguous(input);
    assert_contiguous(out);

    DLTensor &input_tensor = input->dl_tensor;
    DLTensor &out_tensor = out->dl_tensor;

    if(input_tensor.ndim != 3 || out_tensor.ndim != 3) {
        throw std::invalid_argument("need 3D input and output tensors");
    }

    int C = input_tensor.shape[0];
    int H = input_tensor.shape[1];
    int W = input_tensor.shape[2];

    if(!array_equals(0, 3, input_tensor.shape, out_tensor.shape)) {
        throw std::invalid_argument("input and output shapes need to be equal");
    }

    int ksize = (int)(sigma * 6.6f - 2.3f);
    // required kernel size is <= 1, so blur would have no effect
    if(ksize <= 1) {
        copy(input, out);
    }
    else {
        ksize = max(3, ksize | 1);
        const int shared_size = BLOCKSIZE + ksize - 1;
        int shared_memory;
        dim3 grid_dim(1, (W+BLOCKSIZE-1)/BLOCKSIZE, (H+BLOCKSIZE-1)/BLOCKSIZE);
        dim3 block_dim(BLOCKSIZE, BLOCKSIZE, 1);
        DISPATCH(input_tensor.dtype, "gaussian_blur_single_kernel", ([&] {
            shared_memory = (ksize/2+1) * sizeof(temp_t)
                          + shared_size * shared_size * sizeof(temp_t)
                          + BLOCKSIZE * shared_size * sizeof(temp_t);

            if(shared_memory > MAX_SMEM_PER_BLOCK) {
                throw std::invalid_argument("sigma must be <= 30");
            }
            gaussian_blur_single_kernel<scalar_t, temp_t>
            <<<grid_dim, block_dim, shared_memory, current_stream>>>(
                (scalar_t*)input->ptr(),
                (scalar_t*)out->ptr(),
                sigma, ksize, ksize/2, C, H, W, H*W, H*W
            );
        }));
        CUDA(hipGetLastError());
        // mark tensors as in use
        input->record();
        out->record();
    }

    return retval;
}


// namespace augpy
}
