#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "dispatch.h"
#include "saturate_cast.cuh"
#include "elementwise.cuh"
#include <tuple>
#include <algorithm>


namespace augpy {


CudaTensor* make_result_tensor(CudaTensor* tensor, CudaTensor** result) {
    if (!tensor) {
        throw std::invalid_argument("argument 1 may not be None");
    }
    DLTensor t = tensor->dl_tensor;
    CudaTensor* retval = nullptr;
    if (!*result) {
        *result = retval = new CudaTensor(t.shape, t.ndim, dldtype_uint8, t.ctx.device_id);
    }
    return retval;
}


CudaTensor* make_result_tensor(CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor** result) {
    if (!tensor1 || !tensor2) {
        throw std::invalid_argument("argument 1 and 2 may not be None");
    }
    DLTensor t1 = tensor1->dl_tensor;
    DLTensor t2 = tensor2->dl_tensor;
    check_same_dtype_device(t1, t2);
    CudaTensor* retval = nullptr;
    if (!*result) {
        int ndim;
        ndim_array shape;
        calculate_broadcast_output_shape(t1, t2, ndim, shape);
        *result = retval = new CudaTensor(shape.ptr(), ndim, dldtype_uint8, t1.ctx.device_id);
    }
    return retval;
}


template <typename scalar_t>
__device__ __forceinline__ void __lt(
        array<tensor_param, 2> tensors,
        scalar_t scalar
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        (*reinterpret_cast<scalar_t*>(tensors[1].ptr) < scalar);
}


template <typename scalar_t>
__device__ __forceinline__ void __le(
        array<tensor_param, 2> tensors,
        scalar_t scalar
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) <= scalar;
}


template <typename scalar_t>
__device__ __forceinline__ void __gt(
        array<tensor_param, 2> tensors,
        scalar_t scalar
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        (*reinterpret_cast<scalar_t*>(tensors[1].ptr) > scalar);
}


template <typename scalar_t>
__device__ __forceinline__ void __ge(
        array<tensor_param, 2> tensors,
        scalar_t scalar
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) >= scalar;
}


template <typename scalar_t>
__device__ __forceinline__ void __eq(
        array<tensor_param, 2> tensors,
        scalar_t scalar
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) == scalar;
}


template<typename scalar_t>
__device__ __forceinline__ void __lt_tensor(
        array<tensor_param, 3> tensors,
        uint8_t nothing
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) <
        *reinterpret_cast<scalar_t*>(tensors[2].ptr);
}


template<typename scalar_t>
__device__ __forceinline__ void __le_tensor(
        array<tensor_param, 3> tensors,
        uint8_t nothing
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) <
        *reinterpret_cast<scalar_t*>(tensors[2].ptr);
}


template<typename scalar_t>
__device__ __forceinline__ void __eq_tensor(
        array<tensor_param, 3> tensors,
        uint8_t nothing
){
    *reinterpret_cast<uint8_t*>(tensors[0].ptr) = 
        *reinterpret_cast<scalar_t*>(tensors[1].ptr) ==
        *reinterpret_cast<scalar_t*>(tensors[2].ptr);
}


CudaTensor* lt_scalar(
        CudaTensor* tensor,
        double scalar,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor, &result);
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "lt_scalar", ([&] {
        scalar_t casted;
        saturate_cast<double, scalar_t>(scalar, &casted);
        retval = elementwise_function<2, scalar_t, __lt<scalar_t>>(
            tensors, casted, blocks_per_sm, num_threads, false
        );
    }));
    return result;
}


CudaTensor* le_scalar(
        CudaTensor* tensor,
        double scalar,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor, &result);
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "lt_scalar", ([&] {
        scalar_t casted;
        saturate_cast<double, scalar_t>(scalar, &casted);
        retval = elementwise_function<2, scalar_t, __le<scalar_t>>(
            tensors, casted, blocks_per_sm, num_threads, false
        );
    }));
    return result;
}


CudaTensor* gt_scalar(
        CudaTensor* tensor,
        double scalar,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor, &result);
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "lt_scalar", ([&] {
        scalar_t casted;
        saturate_cast<double, scalar_t>(scalar, &casted);
        retval = elementwise_function<2, scalar_t, __gt<scalar_t>>(
            tensors, casted, blocks_per_sm, num_threads, false
        );
    }));
    return result;
}


CudaTensor* ge_scalar(
        CudaTensor* tensor,
        double scalar,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor, &result);
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "lt_scalar", ([&] {
        scalar_t casted;
        saturate_cast<double, scalar_t>(scalar, &casted);
        retval = elementwise_function<2, scalar_t, __ge<scalar_t>>(
            tensors, casted, blocks_per_sm, num_threads, false
        );
    }));
    return result;
}


CudaTensor* eq_scalar(
        CudaTensor* tensor,
        double scalar,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor, &result);
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "lt_scalar", ([&] {
        scalar_t casted;
        saturate_cast<double, scalar_t>(scalar, &casted);
        retval = elementwise_function<2, scalar_t, __eq<scalar_t>>(
            tensors, casted, blocks_per_sm, num_threads, false
        );
    }));
    return result;
}


CudaTensor* lt_tensor(
        CudaTensor* tensor1,
        CudaTensor* tensor2,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor1, tensor2, &result);
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "add_scaled_tensor", ([&] {
        retval = elementwise_function<3, uint8_t, __lt_tensor<scalar_t>>(
            tensors, 0, blocks_per_sm, num_threads, false
        );
    }));
    return retval;
}


CudaTensor* le_tensor(
        CudaTensor* tensor1,
        CudaTensor* tensor2,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor1, tensor2, &result);
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "add_scaled_tensor", ([&] {
        retval = elementwise_function<3, uint8_t, __le_tensor<scalar_t>>(
            tensors, 0, blocks_per_sm, num_threads, false
        );
    }));
    return retval;
}


CudaTensor* eq_tensor(
        CudaTensor* tensor1,
        CudaTensor* tensor2,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = make_result_tensor(tensor1, tensor2, &result);
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "add_scaled_tensor", ([&] {
        retval = elementwise_function<3, uint8_t, __eq_tensor<scalar_t>>(
            tensors, 0, blocks_per_sm, num_threads, false
        );
    }));
    return retval;
}


// namespace augpy
}
