#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor.h"
#include "dispatch.h"
#include "saturate_cast.cuh"
#include "elementwise.cuh"
#include <tuple>


template<typename scalar_t>
__device__ __forceinline__ scalar_t generic_fma(scalar_t x, scalar_t y, scalar_t z);


template<>
__device__ __forceinline__ float generic_fma(float x, float y, float z){
    return fmaf(x,y,z);
}


template<>
__device__ __forceinline__ double generic_fma(double x, double y, double z){
    return fma(x,y,z);
}


namespace augpy {


template <typename scalar_t, typename sscalar_t, typename temp_t>
__device__ __forceinline__ void fma_function(
        array<tensor_param, 3> tensors,
        temp_t scalar
){
    saturate_cast<temp_t, scalar_t>(generic_fma(
        scalar,
        (temp_t) *reinterpret_cast<sscalar_t*>(tensors[1].ptr),
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[2].ptr)
    ), reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


CudaTensor* fma(
        double scalar,
        CudaTensor* tensor1,
        CudaTensor* tensor2,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor2->dl_tensor.dtype, "fma", ([&] {
        ASSERT_TRUE(dldatatype_equals(get_dldatatype<sscalar_t>(), tensor1->dl_tensor.dtype),
                    "tensor1 dtype must be signed version of tensor2 dtype");
        if (result) {
            ASSERT_TRUE(dldatatype_equals(tensor2->dl_tensor.dtype, tensor2->dl_tensor.dtype),
                        "tensor2 dtype must same as result tensor");
        }
        retval = elementwise_function<3, temp_t, fma_function<scalar_t, sscalar_t, temp_t>>(
            tensors,
            (temp_t) scalar,
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


template <typename scalar_t, typename temp_t>
__device__ __forceinline__ void __add_scaled(
        array<tensor_param, 2> tensors,
        std::tuple<temp_t, temp_t> scalars
){
    saturate_cast<temp_t, scalar_t>(generic_fma(
        std::get<1>(scalars),
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr),
        std::get<0>(scalars)
    ), reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


CudaTensor* add_scaled(
        double alpha,
        double beta,
        CudaTensor* tensor,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "add_scaled", ([&] {
        retval = elementwise_function<2, std::tuple<temp_t, temp_t>, __add_scaled<scalar_t, temp_t>>(
            tensors,
            std::make_tuple((temp_t)alpha, (temp_t)beta),
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


template<typename scalar_t, typename temp_t>
__device__ __forceinline__ void __add_scaled_tensor(
        array<tensor_param, 3> tensors,
        std::tuple<temp_t> scalars
){
    saturate_cast<temp_t, scalar_t>(generic_fma(
        std::get<0>(scalars),
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr),
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[2].ptr)
    ), reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


CudaTensor* add_scaled_tensor(
        double alpha,
        CudaTensor* tensor1,
        CudaTensor* tensor2,
        CudaTensor* result,
        unsigned int blocks_per_sm,
        unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "add_scaled_tensor", ([&] {
        retval = elementwise_function<3, std::tuple<temp_t>,
                                      __add_scaled_tensor<scalar_t, temp_t>>(
            tensors,
            std::make_tuple((temp_t)alpha),
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


template<typename scalar_t, typename temp_t>
__device__ __forceinline__ void __mul_scaled_tensor(
        array<tensor_param, 3> tensors,
        std::tuple<temp_t> scalars
){
    saturate_cast<temp_t, scalar_t>(generic_fma(
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr),
        (temp_t) *reinterpret_cast<scalar_t*>(tensors[2].ptr),
        std::get<0>(scalars)
    ), reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


CudaTensor* mul_scaled_tensor(
        double alpha, CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "mul_scaled_tensor", ([&] {
        retval = elementwise_function<3, std::tuple<temp_t>,
                                      __mul_scaled_tensor<scalar_t, temp_t>>(
            tensors,
            std::make_tuple((temp_t)alpha),
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


template<typename scalar_t, typename temp_t>
__device__ __forceinline__ void __rdiv_scaled(
        array<tensor_param, 2> tensors,
        std::tuple<temp_t, temp_t> scalars
){
    saturate_cast<temp_t, scalar_t>(
        std::get<0>(scalars)
        + std::get<1>(scalars) / (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr),
        reinterpret_cast<scalar_t*>(tensors[0].ptr)
    );
}


CudaTensor* rdiv_scaled(
        double alpha, double beta, CudaTensor* tensor, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor);
    DISPATCH(tensor->dl_tensor.dtype, "rdiv_scaled", ([&] {
        retval = elementwise_function<2, std::tuple<temp_t, temp_t>,
                                      __rdiv_scaled<scalar_t, temp_t>>(
            tensors,
            std::make_tuple((temp_t)alpha, (temp_t)beta),
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


template<typename scalar_t, typename temp_t>
__device__ __forceinline__ void __div_scaled_tensor(
        array<tensor_param, 3> tensors,
        std::tuple<temp_t, temp_t> scalars
){
    saturate_cast<temp_t, scalar_t>(
        generic_fma(
            std::get<1>(scalars),
            (temp_t) *reinterpret_cast<scalar_t*>(tensors[1].ptr),
            std::get<0>(scalars)
        ) / (temp_t) *reinterpret_cast<scalar_t*>(tensors[2].ptr),
    reinterpret_cast<scalar_t*>(tensors[0].ptr));
}


CudaTensor* div_scaled_tensor(
        double alpha, double beta, CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    CudaTensor* retval = NULL;
    auto tensors = make_array(result, tensor1, tensor2);
    DISPATCH(tensor1->dl_tensor.dtype, "rdiv_scaled", ([&] {
        retval = elementwise_function<3, std::tuple<temp_t, temp_t>,
                                      __div_scaled_tensor<scalar_t, temp_t>>(
            tensors,
            std::make_tuple((temp_t)alpha, (temp_t)beta),
            blocks_per_sm,
            num_threads
        );
    }));
    return retval;
}


CudaTensor* add_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled(scalar, 1.0, tensor, result,
                      blocks_per_sm, num_threads);
}


CudaTensor* sub_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled(-scalar, 1.0, tensor, result,
                      blocks_per_sm, num_threads);
}


CudaTensor* rsub_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled(scalar, -1.0, tensor, result,
                      blocks_per_sm, num_threads);
}


CudaTensor* mul_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled(0.0, scalar, tensor, result,
                      blocks_per_sm, num_threads);
}


CudaTensor* div_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled(0.0, 1.0/scalar, tensor, result,
                      blocks_per_sm, num_threads);
}


CudaTensor* rdiv_scalar(
        CudaTensor* tensor, double scalar, CudaTensor* result,
        unsigned int blocks_per_sm, unsigned int num_threads
){
    return rdiv_scaled(0.0, scalar, tensor, result,
                       blocks_per_sm, num_threads);
}


CudaTensor* add_tensor(
    CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
    unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled_tensor(1.0, tensor1, tensor2, result,
                             blocks_per_sm, num_threads);
}


CudaTensor* sub_tensor(
    CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
    unsigned int blocks_per_sm, unsigned int num_threads
){
    return add_scaled_tensor(-1.0, tensor2, tensor1, result,
                             blocks_per_sm, num_threads);
}


CudaTensor* mul_tensor(
    CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
    unsigned int blocks_per_sm, unsigned int num_threads
){
    return mul_scaled_tensor(0.0, tensor1, tensor2, result,
                             blocks_per_sm, num_threads);
}


CudaTensor* div_tensor(
    CudaTensor* tensor1, CudaTensor* tensor2, CudaTensor* result,
    unsigned int blocks_per_sm, unsigned int num_threads
){
    return div_scaled_tensor(0.0, 1.0, tensor1, tensor2, result,
                             blocks_per_sm, num_threads);
}


// namespace augpy
}
