#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "core.h"
#include "tensor.h"
#include "gamma.h"
#include "dispatch.h"


namespace augpy {


__device__ __forceinline__ float get_contrast_lookup(float a, float b, float w){
    float lut = 1 / (1 + expf(a+w*(b-a)));
    float max = 0.9981282f;
    float min = 0.0018718f;
    return (lut-min)/(max-min);
}

__device__ __forceinline__ float get_contrast_lookup_negative(float a, float b, float w){
    float lut = 2 * atanh(2 * (a+w*(b-a)) - 1);
    float max = 5.69603851f;
    float min = -5.69603851f;
    return (lut-min)/(max-min);
}


template <typename scalar_t, typename temp_t>
__global__ void gamma_kernel(
        scalar_t* src,
        scalar_t* dst,
        const float* const gamma_grays,
        const float* const gamma_colors,
        const float* const contrasts,
        const size_t N,
        const size_t C,
        const size_t count,
        const temp_t max_value,
        const unsigned int values_per_thread
){
    // contrast and gray gamma for current image
    const float contrast = contrasts[blockIdx.x];
    const float gray = gamma_grays[blockIdx.x];
    const temp_t one_by = 1/(temp_t)max_value;
    // first pixel in current image for this thread
    size_t idx = blockIdx.y * blockDim.x * values_per_thread + threadIdx.x;
    size_t last_idx = min(idx + blockDim.x * values_per_thread, C*count);
    // advanced src and dst to first pixel in image
    src += blockIdx.x * C * count;
    dst += blockIdx.x * C * count;
    // C is not a valid channel index,
    // so this will trigger loading of gamma_colors value
    size_t last_c = C;
    float gamma_color;
    for( ; idx<last_idx; idx+=blockDim.x){
        size_t c = idx / count;
        if (c != last_c) {
            gamma_color = gamma_colors[C*blockIdx.x+c];
        }
        last_c = c;
        scalar_t color = src[idx];
        temp_t fraction = (temp_t)color * one_by;
        temp_t baselookup = fraction;
        temp_t contrastlookup = 0;
        temp_t base = 0;
        if(contrast < 0){
            contrastlookup = get_contrast_lookup_negative(3.348e-03f, 1-3.348e-03f, fraction);
            base = (1+contrast)*baselookup - contrast*contrastlookup;
        }
        else{
            contrastlookup = get_contrast_lookup(6.279f, -6.279f, fraction);
            base = (1-contrast)*baselookup + contrast*contrastlookup;
        }
        dst[idx] = powf(base, gray*gamma_color)*max_value;
    }
}


template<typename scalar_t, typename temp_t, unsigned short VALUE_LIMIT, unsigned char MAX_CHANNELS>
__global__ void gamma_kernel_lut(
        scalar_t* src,
        scalar_t* dst,
        const float* gamma_grays,
        const float* gamma_colors,
        const float* contrasts,
        const size_t C,
        const size_t count,
        const scalar_t max_value,
        const unsigned int values_per_thread
){
    // contrast and gray gamma for current image
    const float gray = gamma_grays[blockIdx.x];
    const float contrast = contrasts[blockIdx.x];
    const temp_t one_by = (temp_t)1.0 / (temp_t)max_value;
    // prepare lookup table in shared memory
    __shared__ scalar_t lut[VALUE_LIMIT*MAX_CHANNELS];
    for (size_t c=0; c<C; ++c) {
        float gamma_color = gamma_colors[C * blockIdx.x + c];
        unsigned short lidx = threadIdx.x;
        for ( ; lidx<=max_value; lidx+=blockDim.x) {
            temp_t fraction = (temp_t)lidx * one_by;
            if(contrast < 0){
                fraction = (1 + contrast) * fraction
                         - contrast * get_contrast_lookup_negative(3.348e-03, 1-3.348e-03, fraction);
            }
            else{
                fraction = (1 - contrast) * fraction
                         + contrast * get_contrast_lookup(6.279, -6.279, fraction);
            }
            lut[c*VALUE_LIMIT+lidx] = powf(fraction, gray * gamma_color) * max_value;
        }
    }

    __syncthreads();

    // first pixel in current image for this thread
    size_t idx = blockIdx.y * blockDim.x * values_per_thread + threadIdx.x;
    size_t last_idx = min(idx + blockDim.x * values_per_thread, C*count);
    // advance src to first pixel for this thread
    src += blockIdx.x * C * count;
    dst += blockIdx.x * C * count;
    // loop until last index
    for( ; idx<last_idx; idx+=blockDim.x){
        dst[idx] = lut[VALUE_LIMIT * (idx / count) + min(src[idx], max_value)];
    }
}


CudaTensor* add_gamma(
    CudaTensor* imtensor,
    CudaTensor* gammagrays,
    CudaTensor* gammacolors,
    CudaTensor* contrasts,
    double max_value,
    CudaTensor* out
){
    assert_contiguous(imtensor);
    assert_contiguous(gammagrays);
    assert_contiguous(gammacolors);
    assert_contiguous(contrasts);

    CudaTensor* retval = NULL;
    if (!out) {
        out = retval = empty_like(imtensor);
    }

	DLTensor &im_tensor = imtensor->dl_tensor;
    int64_t N = im_tensor.shape[0];
    int64_t C = im_tensor.shape[1];
    int64_t H = im_tensor.shape[2];
    int64_t W = im_tensor.shape[3];

    check_same_dtype_device(im_tensor, out->dl_tensor);

    unsigned int num_threads = 0;
    calc_threads(num_threads, im_tensor.ctx.device_id);
    hipDeviceProp_t props = get_device_properties(im_tensor.ctx.device_id);
    unsigned int blocks_per_sm = 8;
    unsigned int num_blocks = ceil_div(props.multiProcessorCount * blocks_per_sm, N);
    int64_t count = H*W;
    int64_t vpt = ceil_div(C*count, num_blocks * num_threads);
    unsigned int values_per_thread = min(
        (int64_t)std::numeric_limits<unsigned int>().max(),
        vpt
    );
    num_blocks = ceil_div(ceil_div(C*count, values_per_thread), num_threads);
    dim3 grid(N, num_blocks);

    if(im_tensor.dtype.code == kDLUInt && im_tensor.dtype.bits == 8 && C <= 4 && max_value <= 256){
        gamma_kernel_lut<unsigned char, float, 256, 4>
        <<<grid, num_threads, 256*4, current_stream>>>(
            (unsigned char*)imtensor->ptr(),
            (unsigned char*)out->ptr(),
            (float*)gammagrays->ptr(),
            (float*)gammacolors->ptr(),
            (float*)contrasts->ptr(),
            C, count, (unsigned char)max_value, values_per_thread
        );
    }
    else{
        DISPATCH(im_tensor.dtype, "add_gamma_kernel_dynamic", ([&] {
            gamma_kernel<scalar_t, temp_t>
            <<<grid, num_threads, 0, current_stream>>>(
                (scalar_t*)imtensor->ptr(),
                (scalar_t*)out->ptr(),
                (float*)gammagrays->ptr(),
                (float*)gammacolors->ptr(),
                (float*)contrasts->ptr(),
                N, C, count, (temp_t)max_value, values_per_thread
            );
        }));
    }

    CUDA(hipGetLastError());

    // mark tensors as in use
    imtensor->record();
    gammagrays->record();
    gammacolors->record();
    contrasts->record();
    out->record();

    return retval;
}


// namespace augpy
}
