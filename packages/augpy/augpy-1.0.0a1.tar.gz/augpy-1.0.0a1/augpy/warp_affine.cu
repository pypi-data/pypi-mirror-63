#include "hip/hip_runtime.h"
#include "warp_affine.h"
#include "hip/hip_runtime.h"
#include "saturate_cast.cuh"
#include "exception.h"
#include "core.h"
#include "tensor.h"
#include "dispatch.h"
#include <iostream>


#ifndef _USE_MATH_DEFINES
#define _USE_MATH_DEFINES
#endif
#include <math.h>


namespace py = pybind11;


namespace augpy {


#define MAX_THREADS 128
#define CHANNEL_BLOCK_SIZE 3
#define deg2rad(angle) angle*M_PI/180
#define MAX_SUPERSAMPLING 3
#define BLOCK_SIZE 16


int make_affine_matrix(
        py::buffer out,
        size_t source_height,
        size_t source_width,
        size_t target_height,
        size_t target_width,
        float angle,
        float scale,
        float aspect,
        float shifty,
        float shiftx,
        float sheary,
        float shearx,
        bool hmirror,
        bool vmirror,
        WarpScaleMode scale_mode,
        int max_supersampling
) {
    if (scale == 0) {
        throw std::invalid_argument("scale must be != 0");
    }
    if (aspect == 0) {
        throw std::invalid_argument("aspect must be != 0");
    }

    py::buffer_info out_info = out.request();
    py::gil_scoped_release release;

    if (!check_contiguous(out_info)) {
        throw std::invalid_argument("need contiguous matrix");
    }
    if (out_info.ndim != 2 || out_info.shape[0] < 2 || out_info.shape[1]!=3) {
        throw std::invalid_argument("need 2x3 matrix");
    }
    if (out_info.format.compare("f") != 0) {
        throw std::invalid_argument("matrix must be float32");
    }

    float* outp = (float*) out_info.ptr;

    double sw = (double)source_width / 2.0;
    double sh = (double)source_height / 2.0;
    double tw = (double)target_width / 2.0;
    double th = (double)target_height / 2.0;

    double r;
    if (scale_mode == WARP_SCALE_LONGEST)
        r = max(sh / th, sw / tw) / scale;
    else if (scale_mode == WARP_SCALE_SHORTEST)
        r = min(sh / th, sw / tw) / scale;
    else
        throw std::invalid_argument("unknown scale mode");

    double wdelta = max(sw - tw*r, tw*r - sw);
    double hdelta = max(sh - th*r, th*r - sh);

    double rad = deg2rad(-angle);
    double cos_rad = cos(rad);
    double sin_rad = sin(rad);

    double p1 = -tw - shearx * th;
    double p2 = -tw * sheary - th;
    double p3 = r * aspect * (hmirror ? -1 : 1);
    double p4 = r / aspect * (vmirror ? -1 : 1);

    outp[0] = cos_rad * p3 + sin_rad * p4 * sheary;
    outp[1] = cos_rad * p3 * shearx + sin_rad * p4;
    outp[2] = cos_rad * p3 * p1 + sin_rad * p4 * p2
            + sw + wdelta * shiftx;

    outp[3] = cos_rad * p4 * sheary - sin_rad * p3;
    outp[4] = cos_rad * p4 - sin_rad * p3 * shearx;
    outp[5] = cos_rad * p4 * p2 - sin_rad * p3 * p1
            + sh + hdelta * shifty;

    int supersampling = max(1, min(max_supersampling, (int) ceil(r)));
    return supersampling;
}


#define warp(x,y,xnew,ynew,m0,m1,m2,m3,m4,m5) { \
    xnew = m0*x+m1*y+m2; \
    ynew = m3*x+m4*y+m5; \
}
#define index_hwc(x,y,c,src_W,src_C) \
    (size_t)(y)*src_W*src_C + (size_t)(x)*src_C + c
#define index_chw(x,y,c,src_H,src_W) \
    c*src_H*src_W + (size_t)(y)*src_W + (size_t)(x)
#define check_index(v, vmax) \
    v>=0 && v<vmax
#define check_floor(v, vmax) \
    v>=0 && v<vmax
#define check_ceil(v, vmax) \
    v>-1 && v<=vmax-1
#define interpolate(x, y, ul, ur, dl, dr) \
    y * (x*ur + (1-x)*ul) + (1-y)*(x*dr + (1-x)*dl)


template <typename scalar_t, typename temp_t>
__global__ void warp_kernel(
        const scalar_t* const src,
        const int src_H, const int src_W, const int src_C,
        scalar_t* dst,
        const int dst_C, const int dst_H, const int dst_W,
        const scalar_t* const background,
        const float m0,
        const float m1,
        const float m2,
        const float m3,
        const float m4,
        const float m5,
        const float offset,
        const float step,
        const temp_t norm
){
    const int block = blockIdx.x;
    const int id = blockIdx.y * blockDim.x + threadIdx.x;
    const int idx = id % dst_W;
    const int idy = id / dst_W;
    const int first_channel = block * CHANNEL_BLOCK_SIZE;
    const int last_channel = min(first_channel+CHANNEL_BLOCK_SIZE, src_C);
    const int n_channels = last_channel - first_channel;
    if(id >= dst_H*dst_W) return;
    temp_t color[CHANNEL_BLOCK_SIZE];
    for(int i=0; i<n_channels; i++){
        color[i]=0;
    }
    temp_t bg[CHANNEL_BLOCK_SIZE];
    for(int c=0; c<n_channels; c++){
        bg[c] = background[c+first_channel];
    }
    float ymin = (float)idy - offset;
    float ymax = (float)idy + offset + 1e-5f;
    float xmin = (float)idx - offset;
    float xmax = (float)idx + offset + 1e-5f;
    for(float posy=ymin; posy<ymax; posy+=step){
        for(float posx=xmin; posx<xmax; posx+=step){
            float newx, newy;
            warp(posx, posy, newx, newy, m0, m1, m2, m3, m4, m5);
            float floor_x = floor(newx);
            float floor_y = floor(newy);
            float ceil_x = ceil(newx);
            float ceil_y = ceil(newy);
            float rx = newx - floor_x;
            float ry = ceil_y - newy;
            bool floor_x_ok = check_index(floor_x, src_W);
            bool floor_y_ok = check_index(floor_y, src_H);
            bool ceil_x_ok = check_index(ceil_x, src_W);
            bool ceil_y_ok = check_index(ceil_y, src_H);
            for(int c=first_channel; c<last_channel; c++){
                temp_t back = bg[c-first_channel];
                float ul = floor_x_ok && floor_y_ok ?
                    src[index_hwc(floor_x,floor_y,c,src_W,src_C)] : back;
                float dl = floor_x_ok && ceil_y_ok ?
                    src[index_hwc(floor_x,ceil_y,c,src_W,src_C)] : back;
                float ur = ceil_x_ok && floor_y_ok ?
                    src[index_hwc(ceil_x,floor_y,c,src_W,src_C)] : back;
                float dr = ceil_x_ok && ceil_y_ok ?
                    src[index_hwc(ceil_x,ceil_y,c,src_W,src_C)] : back;
                color[c-first_channel] += interpolate(rx, ry, ul, ur, dl, dr);
            }
        }
    }
    for(size_t c=first_channel; c<last_channel; c++){
        saturate_cast<temp_t, scalar_t>(
            color[c-first_channel] * norm,
            &dst[index_chw(idx,idy,c,dst_H,dst_W)]
        );
    }
}


void warp_affine(
        CudaTensor* src,
        CudaTensor* dst,
        py::buffer matrix,
        CudaTensor* background,
        int supersampling
){
    py::buffer_info matrix_info = matrix.request();

    py::gil_scoped_release release;

    check_tensor(src);
    check_tensor(dst);
    check_tensor(background);

    DLTensor& st = src->dl_tensor;
    DLTensor& dt = dst->dl_tensor;
    DLTensor& bt = background->dl_tensor;

    if (st.ndim != 3 || dt.ndim != 3) {
        throw std::invalid_argument("need 3D DLTensors for src and dst");
    }
    if (st.shape[2] != dt.shape[0]) {
        throw std::invalid_argument("src shape[2] and dst shape[0] must match");
    }
    if (!check_contiguous(matrix_info)) {
        throw std::invalid_argument("need contiguous matrix");
    }
    if (matrix_info.ndim != 2 || matrix_info.shape[0] < 2 || matrix_info.shape[1]!=3){
        throw std::invalid_argument("need 2x3 matrix");
    }
    if (matrix_info.format.compare("f") != 0) {
        throw std::invalid_argument("matrix must be float32");
    }
    if (bt.shape[0] != dt.shape[0]) {
        throw std::invalid_argument("background must match channels");
    }

    const float* matrixp = (const float*) matrix_info.ptr;
    const size_t channel_blocks = ceil_div(dt.shape[0], CHANNEL_BLOCK_SIZE);
    const float step = 1.0f / (float) supersampling;
    const float offset = (float)(supersampling - 1) * step / 2.0f;
    const double norm = 1.0 / (double)(supersampling*supersampling);
    int num_blocks = ceil_div(dt.shape[1] * dt.shape[2], MAX_THREADS);
    dim3 grid(channel_blocks, num_blocks, 1);

    DISPATCH(st.dtype, "warp_kernel", ([&] {
        warp_kernel<scalar_t, temp_t>
        <<<grid, MAX_THREADS, 0, current_stream>>>(
                (const scalar_t*) src->ptr(),
                st.shape[0], st.shape[1], st.shape[2],
                (scalar_t*) dst->ptr(),
                dt.shape[0], dt.shape[1], dt.shape[2],
                (const scalar_t*) background->ptr(),
                matrixp[0],
                matrixp[1],
                matrixp[2],
                matrixp[3],
                matrixp[4],
                matrixp[5],
                offset,
                step,
                norm
        ); })
    );

    CUDA(hipGetLastError());
    // mark tensors as in use
    src->record();
    dst->record();
    background->record();
}


// namespace augpy
}
